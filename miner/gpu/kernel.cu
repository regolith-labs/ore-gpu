#include "hip/hip_runtime.h"
#include <equix.h>
#include <hashx.h>
#include <rNdN.h>

__constant__ hashx_sass_binary* d_binaries; // Cache of compiled SASS
__constant__ uint32_t max_binaries;

__global__ void equix_gpu_solver(
    const equix_challenge* challenges,
    equix_solution* solutions,
    uint32_t num_challenges
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= num_challenges) return;

    // Stage 0: Generate HashX function
    hashx_sass_binary* bin = &d_binaries[idx % max_binaries];
    uint64_t nonce = challenges[idx].nonce;
    uint64_t r[8];

    // Execute compiled HashX via SASS
    asm volatile (
        "{\n"
        ".reg .b32 %r_hashid;\n"
        "mov.u32 %r_hashid, %0;\n"
        "call.uni (%r_hashid), _hashx_sass_entry;\n"
        "}\n" 
        :: "r"(bin->id) 
        : "memory"
    );

    // Stage 1: Build hash table
    __shared__ struct {
        uint64_t indices[1 << 18]; // 1MB shared mem
        uint32_t count;
    } stage1;

    if(threadIdx.x == 0) stage1.count = 0;
    __syncthreads();

    for(uint32_t i = 0; i < (1 << 16); i++) {
        uint64_t h = hashx_exec_sass(bin, i);
        uint32_t bucket = (h >> 40) & 0xfff; // 12-bit bucket
        
        uint32_t pos = atomicAdd(&stage1.count, 1);
        stage1.indices[pos] = (bucket << 40) | (i & 0xfffff);
    }
    __syncthreads();

    // Stage 2: Find collisions
    __shared__ struct {
        uint64_t pairs[1 << 17];
        uint32_t count;
    } stage2;

    if(threadIdx.x == 0) stage2.count = 0;
    __syncthreads();

    for(uint32_t i = threadIdx.x; i < stage1.count; i += blockDim.x) {
        uint64_t entry = stage1.indices[i];
        uint32_t bucket = entry >> 40;
        
        // Linear probe matching (optimized for warp efficiency)
        for(uint32_t j = i + 1; j < stage1.count; j++) {
            if((stage1.indices[j] >> 40) == bucket) {
                uint32_t pos = atomicAdd(&stage2.count, 1);
                stage2.pairs[pos] = (entry << 20) | (stage1.indices[j] & 0xfffff);
            }
        }
    }
    __syncthreads();

    // Stage 3: Validate solutions
    for(uint32_t i = threadIdx.x; i < stage2.count; i += blockDim.x) {
        uint64_t pair = stage2.pairs[i];
        uint32_t idx1 = (pair >> 20) & 0xfffff;
        uint32_t idx2 = pair & 0xfffff;

        uint64_t h1 = hashx_exec_sass(bin, idx1);
        uint64_t h2 = hashx_exec_sass(bin, idx2);

        if((h1 + h2) & 0xffffffffffc00000) continue; // Partial sum check

        // Store valid solution
        uint32_t sol_idx = atomicAdd(&solutions->count, 1);
        if(sol_idx < EQUIX_MAX_SOLS) {
            solutions->sols[sol_idx].idx[0] = idx1;
            solutions->sols[sol_idx].idx[1] = idx2;
        }
    }
}

// Host-side management
struct GPUSolver {
    hashx_sass_binary* d_binaries;
    uint32_t bin_capacity;
    uint32_t bin_count;

    GPUSolver(size_t max_binaries) : bin_capacity(max_binaries) {
        hipMalloc(&d_binaries, max_binaries * sizeof(hashx_sass_binary));
        hipMemcpyToSymbol(HIP_SYMBOL(::d_binaries), &d_binaries, sizeof(hashx_sass_binary*));
        hipMemcpyToSymbol(HIP_SYMBOL(::max_binaries), &max_binaries, sizeof(uint32_t));
    }

    void add_binary(const hashx_sass_binary& bin) {
        if(bin_count >= bin_capacity) {
            // LRU eviction
            hipMemcpy(&d_binaries[bin_count % bin_capacity], &bin, 
                sizeof(hashx_sass_binary), hipMemcpyHostToDevice);
        } else {
            hipMemcpy(&d_binaries[bin_count], &bin, 
                sizeof(hashx_sass_binary), hipMemcpyHostToDevice);
        }
        bin_count++;
    }

    void solve_batch(const equix_challenge* challenges, equix_solution* solutions, 
                     size_t count) {
        const int threads = 256;
        const int blocks = (count + threads - 1) / threads;
        
        equix_gpu_solver<<<blocks, threads>>>(challenges, solutions, count);
        hipDeviceSynchronize();
    }

    ~GPUSolver() {
        hipFree(d_binaries);
    }
};

// Test Kernel
__global__ void hashx_test_kernel(
    const hashx_sass_binary* bin,
    uint64_t nonce,
    uint8_t* output
) {
    // Execute compiled HashX
    asm volatile (
        "{\n"
        ".reg .b32 %r_hashid;\n"
        "mov.u32 %r_hashid, %0;\n"
        "call.uni (%r_hashid), _hashx_sass_entry;\n"
        "}\n" 
        :: "r"(bin->id) 
        : "memory"
    );

    // Store result
    if(threadIdx.x == 0) {
        *((uint64_t*)output) = nonce * 123; // Example hash computation
    }
}

// Verification Test
bool test_hashx_gpu(const hashx_ctx* cpu_ctx, uint64_t nonce) {
    // 1. Compile HashX program
    HashXCompiler compiler;
    auto sass_bin = compiler.compile(cpu_ctx->program);

    // 2. GPU Memory Setup
    hashx_sass_binary* d_bin;
    uint8_t* d_output;
    hipMalloc(&d_bin, sizeof(hashx_sass_binary));
    hipMalloc(&d_output, HASHX_SIZE);

    hipMemcpy(d_bin, &sass_bin, sizeof(hashx_sass_binary), hipMemcpyHostToDevice);

    // 3. Launch Kernel
    hashx_test_kernel<<<1, 1>>>(d_bin, nonce, d_output);
    hipDeviceSynchronize();

    // 4. Get Result
    uint8_t gpu_hash[HASHX_SIZE];
    hipMemcpy(gpu_hash, d_output, HASHX_SIZE, hipMemcpyDeviceToHost);

    // 5. CPU Verification 
    uint8_t cpu_hash[HASHX_SIZE];
    hashx_exec(cpu_ctx, nonce, cpu_hash);

    // 6. Cleanup
    hipFree(d_bin);
    hipFree(d_output);

    return memcmp(gpu_hash, cpu_hash, HASHX_SIZE) == 0;
} 